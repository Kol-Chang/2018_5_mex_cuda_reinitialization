#include "hip/hip_runtime.h"
#include "mex.h"
#include "mexcudaReinitialization.hpp"
#include <hip/hip_runtime_api.h>

__global__ 
void ExploreIdx()
{
	unsigned int const block_idx = blockIdx.x;
	unsigned int const thread_idx = threadIdx.x;
	//mexPrintf("This is block (%d)", blockIdx.x);
} 

void Reinitialization(void){
	mexPrintf("hello cuda!\n");

	// 
	dim3 const dimBlock(1,1,1);
	dim3 const dimThread(2,1,1);

	mexPrintf("Block dimension (%d)\n", dimBlock.x);

	ExploreIdx<<<dimBlock,dimThread>>>();
}