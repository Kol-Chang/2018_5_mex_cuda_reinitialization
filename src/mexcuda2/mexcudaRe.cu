#include <algorithm>
#include <hip/hip_runtime_api.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"

#include "Re.hpp"

// enumerate input index
enum in_put{
	level_set_function = 0,
	grid_spacing = 1
};

// enumerate output index
enum out_put{
	reinitialized_lsf = 0
};

/**
 * MEX gateway
 */
void mexFunction(int nlhs , mxArray *plhs[], int nrhs, mxArray const * prhs[])
{
	// Initialize the MathWorks GPU API.
    mxInitGPU();

    mxClassID category;

    if(nrhs != 2){
		mexErrMsgIdAndTxt("mexReinitialization:wrong_number_of_inputs",
			"expecting 2 inputs: a 3d array representing level set function and a 1x3 array representing grid spacing");
	}

	// assign level set function
	mxGPUArray const * lsf = mxGPUCreateFromMxArray(prhs[level_set_function]);
	double const *dev_lsf = (double const *)(mxGPUGetDataReadOnly(lsf)); // pointer to input data on device

	mwSize number_of_dimensions;
	const mwSize *dimension_array;
	size_t number_of_elements_lsf;

	category = mxGPUGetClassID(lsf);
	number_of_dimensions = mxGPUGetNumberOfDimensions(lsf);
	dimension_array = mxGPUGetDimensions(lsf);
	number_of_elements_lsf = mxGPUGetNumberOfElements(lsf);

	if (category != mxDOUBLE_CLASS || number_of_dimensions != (mwSize)3 || !mxIsGPUArray(prhs[level_set_function])){
		mexErrMsgIdAndTxt("mexReinitialization:Invalid_Input",
			"Argument %d must be a 3 dimension GPUarray of double precision!",
			level_set_function);
	}
	// finish assigning level set function

	// assign grid spacing array
	double *ds;
	size_t rows, cols;

	category = mxGetClassID(prhs[grid_spacing]);
	rows = mxGetM(prhs[grid_spacing]);
	cols = mxGetN(prhs[grid_spacing]);
	if (category != mxDOUBLE_CLASS || rows != (size_t)1 || cols != (size_t)3){
		mexErrMsgIdAndTxt("mexReinitialization:Invalid_Input",
			"Argument %d must be a 1X3 double array of the grid spacing",
			grid_spacing);
	}
	ds = (double *)mxGetData(prhs[grid_spacing]);
	// finish assigning spacing array


	/* Create a GPUArray to hold the result and get its underlying pointer. */
 	mxGPUArray *re_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);

 	double * dev_re_lsf = (double *)(mxGPUGetData(re_lsf)); // pointer to data on device


    /*
     * workspace gpuArrays
     */
    mxGPUArray *xpr = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_xpr = (double *)(mxGPUGetData(xpr)); // pointer to data on device

 	mxGPUArray *ypf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_ypf = (double *)(mxGPUGetData(ypf)); // pointer to data on device

 	mxGPUArray *zpu = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_zpu = (double *)(mxGPUGetData(zpu)); // pointer to data on device

 	mxGPUArray *new_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_new_lsf = (double *)(mxGPUGetData(new_lsf)); // pointer to data on device

 	mxGPUArray *intermediate_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_intermediate_lsf = (double *)(mxGPUGetData(intermediate_lsf)); // pointer to data on device

 	mxGPUArray *cur_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_cur_lsf = (double *)(mxGPUGetData(cur_lsf)); // pointer to data on device

 	// call the computation routine
 	Reinitialization(dev_re_lsf, dev_lsf, 
 		dev_xpr, dev_ypf, dev_zpu,
 		dev_new_lsf, dev_intermediate_lsf, dev_cur_lsf,
 		number_of_elements_lsf, dimension_array[0], dimension_array[1], dimension_array[2],
 		ds[0], ds[1], ds[2]);



 	/* Wrap the result up as a MATLAB gpuArray for return. */
    //plhs[reinitialized_lsf] = mxGPUCreateMxArrayOnGPU(re_lsf);

	/* Wrap the result up as a MATLAB cpuArray for return. */
	mexPrintf("trying to return a cpu array \n");
    //plhs[reinitialized_lsf] = mxGPUCreateMxArrayOnCPU(cur_lsf);
    plhs[reinitialized_lsf] = mxGPUCreateMxArrayOnCPU(re_lsf);

    mexPrintf("returned a cpu array \n");

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
	mxGPUDestroyGPUArray(lsf);
	mxGPUDestroyGPUArray(re_lsf);
	mxGPUDestroyGPUArray(xpr);
	mxGPUDestroyGPUArray(ypf);
	mxGPUDestroyGPUArray(zpu);
	mxGPUDestroyGPUArray(new_lsf);
	mxGPUDestroyGPUArray(intermediate_lsf);
	mxGPUDestroyGPUArray(cur_lsf);
}



























































