#include "mex.h"
#include "gpu/mxGPUArray.h"

#include "Re.hpp"

// enumerate input index
enum in_put{
	level_set_function = 0,
	grid_spacing = 1
};

// enumerate output index
enum out_put{
	reinitialized_lsf = 0
};

/**
 * MEX gateway
 */
void mexFunction(int nlhs , mxArray *plhs[], int nrhs, mxArray const * prhs[])
{
	// Initialize the MathWorks GPU API.
    mxInitGPU();

    mxClassID category;

    mexPrintf("2nd try ...\n");

    if(nrhs != 2){
		mexErrMsgIdAndTxt("mexReinitialization:wrong_number_of_inputs",
			"expecting 2 inputs");
	}

	// assign level set function
	
	mxGPUArray const * lsf = mxGPUCreateFromMxArray(prhs[level_set_function]);
	double const *dev_lsf = (double const *)(mxGPUGetDataReadOnly(lsf)); // pointer to input data on device

	mwSize number_of_dimensions;
	const mwSize *dimension_array;
	size_t number_of_elements_lsf;

	category = mxGPUGetClassID(lsf);
	number_of_dimensions = mxGPUGetNumberOfDimensions(lsf);
	dimension_array = mxGPUGetDimensions(lsf);
	number_of_elements_lsf = mxGPUGetNumberOfElements(lsf);

	if (category != mxDOUBLE_CLASS || number_of_dimensions != (mwSize)3 || !mxIsGPUArray(lsf)){
		mexErrMsgIdAndTxt("mexReinitialization:Invalid_Input",
			"Argument %d must be a 3 dimension array of double precision!",
			level_set_function);
	}



	//
	mxGPUArray const * ds = mxGPUCreateFromMxArray(prhs[grid_spacing]);
	double const *dev_ds = (double const *)(mxGPUGetDataReadOnly(ds)); 

	/* Create a GPUArray to hold the result and get its underlying pointer. */
 	mxGPUArray *re_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);

 	double * dev_re_lsf = (double *)(mxGPUGetData(re_lsf)); // pointer to data on device


    /*
     * workspace gpuArrays
     */
    mxGPUArray *xpr = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_xpr = (double *)(mxGPUGetData(xpr)); // pointer to data on device

 	mxGPUArray *ypf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_ypf = (double *)(mxGPUGetData(ypf)); // pointer to data on device

 	mxGPUArray *zpu = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_zpu = (double *)(mxGPUGetData(zpu)); // pointer to data on device

 	mxGPUArray *new_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_new_lsf = (double *)(mxGPUGetData(new_lsf)); // pointer to data on device

 	mxGPUArray *intermediate_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_intermediate_lsf = (double *)(mxGPUGetData(intermediate_lsf)); // pointer to data on device

 	mxGPUArray *cur_lsf = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(lsf),
                            				 mxGPUGetDimensions(lsf),
                            				 mxGPUGetClassID(lsf),
                            				 mxGPUGetComplexity(lsf),
                            				 MX_GPU_DO_NOT_INITIALIZE);
 	double * dev_cur_lsf = (double *)(mxGPUGetData(cur_lsf)); // pointer to data on device





 	/* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[reinitialized_lsf] = mxGPUCreateMxArrayOnGPU(re_lsf);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
	mxGPUDestroyGPUArray(lsf);
	mxGPUDestroyGPUArray(ds);
	mxGPUDestroyGPUArray(xpr);
	mxGPUDestroyGPUArray(ypf);
	mxGPUDestroyGPUArray(zpu);
	mxGPUDestroyGPUArray(new_lsf);
	mxGPUDestroyGPUArray(intermediate_lsf);
	mxGPUDestroyGPUArray(cur_lsf);
}



























































